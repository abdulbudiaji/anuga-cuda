
#include <hip/hip_runtime.h>
__global__ void set_boundary_values_from_edges(
        int N,
        int * vol_id,
        int * edge_id,
        double * boundary_values,
        double * edge_values)
{
    const int k = 
            threadIdx.x+threadIdx.y*blockDim.x+
            (blockIdx.x+blockIdx.y*gridDim.x)*blockDim.x*blockDim.y;
    
    if ( k >= N )
        return;

    int id = 3* vol_id[k] + edge_id[k];

    boundary_values[id] = edge_values[id];
}
