
#include <hip/hip_runtime.h>
//#define REARRANGED_DOMAIN 

#define BLOCK_SIZE 960

__global__ void gravity_wb(
        int N,
        double g,
        double * stage_vertex_values, 
        double * stage_edge_values, 
        double * stage_centroid_values, 
        double * bed_edge_values, 
        double * bed_centroid_values, 
        double * vertex_coordinates, 
        double * xmom_explicit_update, 
        double * ymom_explicit_update, 
        double * normals, 
        double * areas, 
        double * edgelengths
        )
{
    const int k = 
            threadIdx.x+threadIdx.y*blockDim.x+
            (blockIdx.x+blockIdx.y*gridDim.x)*blockDim.x*blockDim.y;

    int i;
#ifndef REARRANGED_DOMAIN
    int k3=k*3;
#endif

    double w0, w1, w2, 
           x0, y0, x1, y1, x2, y2,
           avg_h;

    double wx, wy, det,
           hh[3];
    //double sidex, sidey;
    double area, n0, n1, fact;

    __shared__ double sh_data[ BLOCK_SIZE *6];

    if (k >= N)
        return;
#ifndef REARRANGED_DOMAIN
    w0 = stage_vertex_values[k3];
    w1 = stage_vertex_values[k3 + 1];
    w2 = stage_vertex_values[k3 + 2];

    x0 = vertex_coordinates[k*6];
    y0 = vertex_coordinates[k*6 + 1];
    x1 = vertex_coordinates[k*6 + 2];
    y1 = vertex_coordinates[k*6 + 3];
    x2 = vertex_coordinates[k*6 + 4];
    y2 = vertex_coordinates[k*6 + 5];
#else
    w0 = stage_vertex_values[k];
    w1 = stage_vertex_values[k + N];
    w2 = stage_vertex_values[k + 2*N];
    
    x0 = vertex_coordinates[k];
    y0 = vertex_coordinates[k + N];
    x1 = vertex_coordinates[k + 2*N];
    y1 = vertex_coordinates[k + 3*N];
    x2 = vertex_coordinates[k + 4*N];
    y2 = vertex_coordinates[k + 5*N];
#endif

    //_gradient(x0, y0, x1, y1, x2, y2, w0, w1, w2, &wx, &wy);

    det = (y2 - y0)*(x1 - x0) - (y1 - y0)*(x2 - x0);

    wx = (y2 -y0)*(w1 - w0) - (y1 - y0)*(w2 -w0);
    wx /= det;

    wy = (x1 - x0)*(w2 - w0) - (x2 - x0)*(w1 -w0);
    wy /= det;


    avg_h = stage_centroid_values[k] - bed_centroid_values[k];

    xmom_explicit_update[k] += -g * wx * avg_h;
    ymom_explicit_update[k] += -g * wy * avg_h;

#ifndef REARRANGED_DOMAIN
    hh[0] = stage_edge_values[k3] - bed_edge_values[k3];
    hh[1] = stage_edge_values[k3+1] - bed_edge_values[k3+1];
    hh[2] = stage_edge_values[k3+2] - bed_edge_values[k3+2];
#else
    hh[0] = stage_edge_values[k] - bed_edge_values[k];
    hh[1] = stage_edge_values[k+N] - bed_edge_values[k+N];
    hh[2] = stage_edge_values[k+2*N] - bed_edge_values[k+2*N];
#endif

    //sidex = 0.0;
    //sidey = 0.0;
    area = areas[k];

    for ( i = 0 ; i < 3 ; i++ )
    {
#ifndef REARRANGED_DOMAIN
        n0 = normals[k*6 + 2*i];
        n1 = normals[k*6 + 2*i + 1];

        fact =  -0.5 * g * hh[i] * hh[i] * edgelengths[k3 + i];
#else
        n0 = normals[k + 2*i*N];
        n1 = normals[k + (2*i + 1)*N];

        fact =  -0.5 * g * hh[i] * hh[i] * edgelengths[k + i*N];
#endif

        //sidex += fact*n0;
        //sidey += fact*n1;

        sh_data[threadIdx.x + i*blockDim.x] = fact*n0;
        sh_data[threadIdx.x + (i+3)*blockDim.x] = fact*n1;
    }

    //xmom_explicit_update[k] += -sidex / area;
    //ymom_explicit_update[k] += -sidey / area;

    xmom_explicit_update[k] += -(sh_data[threadIdx.x] + sh_data[threadIdx.x + blockDim.x] + sh_data[threadIdx.x+2*blockDim.x]) / area;

    ymom_explicit_update[k] += -(sh_data[threadIdx.x+3*blockDim.x] + sh_data[threadIdx.x + 4*blockDim.x] + sh_data[threadIdx.x+5*blockDim.x]) / area;
}
