
#include <hip/hip_runtime.h>
//#define REARRANGED_DOMAIN 

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 960
#endif

__global__ void gravity_wb(
        int N,
        double g,
        double * stage_vertex_values, 
        double * stage_edge_values, 
        double * stage_centroid_values, 
        double * bed_edge_values, 
        double * bed_centroid_values, 
        double * vertex_coordinates, 
        double * xmom_explicit_update, 
        double * ymom_explicit_update, 
        double * normals, 
        double * areas, 
        double * edgelengths
        )
{
    const int k = 
            threadIdx.x+threadIdx.y*blockDim.x+
            (blockIdx.x+blockIdx.y*gridDim.x)*blockDim.x*blockDim.y;

    int i;
#ifndef REARRANGED_DOMAIN
    int k3=k*3, k6=k*6;
#endif

    double w0, w1, w2, 
           x0, y0, x1, y1, x2, y2,
           avg_h;

    double wx, wy, det,
           hh[3];
    double area, n0, n1, fact;

#ifdef USING_SHARED_MEMORY
    __shared__ double sh_data[ BLOCK_SIZE *6];
#else
    double sidex=0, sidey=0;
#endif
    if (k >= N)
        return;

    avg_h = stage_centroid_values[k] - bed_centroid_values[k];

    avg_h = g * avg_h;

#ifndef REARRANGED_DOMAIN
    w0 = stage_vertex_values[k3];
    w1 = stage_vertex_values[k3 + 1];
    w2 = stage_vertex_values[k3 + 2];

    n0 = w1 - w0;
    n1 = w2 - w0;

    x0 = vertex_coordinates[k6];
    x1 = vertex_coordinates[k6 + 2];
    sidex = x1 - x0;
    wy = sidex*n1;
    x2 = vertex_coordinates[k6 + 4];
    sidey = x2 - x0;
    wy -= sidey*n0;

    y0 = vertex_coordinates[k6 + 1];
    y1 = vertex_coordinates[k6 + 3];
    x0 = y1-y0;
    wx = x0*n1;
    det = x0 *sidey;
    y2 = vertex_coordinates[k6 + 5];
    x1 = y2 - y0;
    wx = x1*n0 - wx;
    det = x1*sidex - det;
    wx /= det;
    wy /= det;

    x0 = wx * avg_h;
    x1 = wy * avg_h;

#else
    w0 = stage_vertex_values[k];
    w1 = stage_vertex_values[k + N];
    w2 = stage_vertex_values[k + 2*N];
    
    n0 = w1 - w0;
    n1 = w2 - w0;

    x0 = vertex_coordinates[k];
    x1 = vertex_coordinates[k + 2*N];
    sidex = x1 - x0;
    wy = sidex*n1;
    x2 = vertex_coordinates[k + 4*N];
    sidey = x2 - x0;
    wy -= sidey*n0;

    
    y0 = vertex_coordinates[k + N];
    y1 = vertex_coordinates[k + 3*N];
    x0 = y1-y0;
    wx = x0*n1;
    y2 = vertex_coordinates[k + 5*N];

    _gradient(x0, y0, x1, y1, x2, y2, w0, w1, w2, &wx, &wy);

    det = (y2 - y0)*(x1 - x0) - (y1 - y0)*(x2 - x0);

    wx = (y2 -y0)*(w1 - w0) - (y1 - y0)*(w2 -w0);
    wx /= det;

    wy = (x1 - x0)*(w2 - w0) - (x2 - x0)*(w1 -w0);
    wy /= det;

    avg_h = stage_centroid_values[k] - bed_centroid_values[k];

    xmom_explicit_update[k] += -g *wx *avg_h;
    ymom_explicit_update[k] += -g *wy *avg_h;
#endif

#ifndef REARRANGED_DOMAIN
    hh[0] = stage_edge_values[k3] - bed_edge_values[k3];
    hh[0] *= -0.5 * g * hh[0];
    hh[1] = stage_edge_values[k3+1] - bed_edge_values[k3+1];
    hh[1] *= -0.5 * g * hh[1];
    hh[2] = stage_edge_values[k3+2] - bed_edge_values[k3+2];
    hh[2] *= -0.5 * g * hh[2];
#else
    hh[0] = stage_edge_values[k] - bed_edge_values[k];
    hh[0] *= -0.5 * g * hh[0];
    hh[1] = stage_edge_values[k+N] - bed_edge_values[k+N];
    hh[1] *= -0.5 * g * hh[1];
    hh[2] = stage_edge_values[k+2*N] - bed_edge_values[k+2*N];
    hh[2] *= -0.5 * g * hh[2];
#endif

    area = areas[k];


#ifndef USING_SHARED_MEMORY
    sidex = 0;
    sidey = 0;
#endif


    for ( i = 0 ; i < 3 ; i++ )
    {
#ifndef REARRANGED_DOMAIN
        n0 = normals[k6 + 2*i];
        n1 = normals[k6 + 2*i + 1];

       // fact =  -0.5 * g * hh[i] * hh[i] * edgelengths[k3 + i];
        fact = hh[i] * edgelengths[k3 + i];
#else
        n0 = normals[k + 2*i*N];
        n1 = normals[k + (2*i + 1)*N];

        fact = hh[i] * edgelengths[k + i*N];
#endif

#ifdef USING_SHARED_MEMORY
        //sh_data[threadIdx.x + i*blockDim.x] = fact*n0;
        //sh_data[threadIdx.x + (i+3)*blockDim.x] = fact*n1;
#else
        sidex += fact*n0;
        sidey += fact*n1;
#endif
    }

    
#ifdef USING_SHARED_MEMORY
    //xmom_explicit_update[k] += -(sh_data[threadIdx.x] + sh_data[threadIdx.x + blockDim.x] + sh_data[threadIdx.x+2*blockDim.x]) / area;

    //ymom_explicit_update[k] += -(sh_data[threadIdx.x+3*blockDim.x] + sh_data[threadIdx.x + 4*blockDim.x] + sh_data[threadIdx.x+5*blockDim.x]) / area;
#else
//    xmom_explicit_update[k] += -sidex / area -g *wx *avg_h;
//    ymom_explicit_update[k] += -sidey / area -g *wy *avg_h;

    sidex /= area;
    sidey /= area;
    x0 = sidex + x0;
    x1 = sidey + x1;
    //w0 = xmom_explicit_update[k] - w0;// - x0;
    xmom_explicit_update[k] -= x0;
    //w1 = ymom_explicit_update[k] - w1;// - x1;
    ymom_explicit_update[k] -= x1;

    //n0 = xmom_explicit_update[k] - (sidex / area +g *wx *avg_h);
    //xmom_explicit_update[k] = n0;
    //n1 = ymom_explicit_update[k] - (sidey / area +g *wy *avg_h);
    //ymom_explicit_update[k] = n1;
#endif
}
