
#include <hip/hip_runtime.h>
//#include<Python.h>
//#include "extrapolate.h"

#define    W = 16
#define STRINGIFY(x) #x
#define TOSTRING(x) STRINGIFY(x)
#define AT __FILE__ ":" TOSTRING(__LINE__)
#define P_ERROR_BUFFER_SIZE 65

__device__ void report_python_error(const char *location, const char *msg)
{

    //    char buf[P_ERROR_BUFFER_SIZE];

    //    snprintf(buf, P_ERROR_BUFFER_SIZE, "Error at %s: %s\n", location, msg);

    //    PyErr_SetString(PyExc_RuntimeError, buf);
}

__device__ int limit_gradient(double *dqv, double qmin, double qmax, double beta_w) {
    // Given provisional jumps dqv from the FV triangle centroid to its
    // vertices and jumps qmin (qmax) between the centroid of the FV
    // triangle and the minimum (maximum) of the values at the centroid of
    // the FV triangle and the auxiliary triangle vertices,
    // calculate a multiplicative factor phi by which the provisional
    // vertex jumps are to be limited

    int i;
    double r = 1000.0, r0 = 1.0, phi = 1.0;
    //static double TINY = 1.0e-100; // to avoid machine accuracy problems.
    double TINY = 1.0e-100; // to avoid machine accuracy problems.
    // FIXME: Perhaps use the epsilon used elsewhere.

    // Any provisional jump with magnitude < TINY does not contribute to
    // the limiting process.
    for (i = 0; i < 3; i++) {
        if (dqv[i]<-TINY)
            r0 = qmin / dqv[i];

        if (dqv[i] > TINY)
            r0 = qmax / dqv[i];

        r = min(r0, r);
    }

    phi = min(r*beta_w, 1.0);
    //for (i=0;i<3;i++)
    dqv[0] = dqv[0] * phi;
    dqv[1] = dqv[1] * phi;
    dqv[2] = dqv[2] * phi;

    return 0;
}

__device__ int find_qmin_and_qmax(double dq0, double dq1, double dq2,
        double *qmin, double *qmax) {
    // Considering the centroid of an FV triangle and the vertices of its
    // auxiliary triangle, find
    // qmin=min(q)-qc and qmax=max(q)-qc,
    // where min(q) and max(q) are respectively min and max over the
    // four values (at the centroid of the FV triangle and the auxiliary
    // triangle vertices),
    // and qc is the centroid
    // dq0=q(vertex0)-q(centroid of FV triangle)
    // dq1=q(vertex1)-q(vertex0)
    // dq2=q(vertex2)-q(vertex0)

    if (dq0 >= 0.0) {
        if (dq1 >= dq2) {
            if (dq1 >= 0.0)
                *qmax = dq0 + dq1;
            else
                *qmax = dq0;

            *qmin = dq0 + dq2;
            if (*qmin >= 0.0) *qmin = 0.0;
        } else {// dq1<dq2
            if (dq2 > 0)
                *qmax = dq0 + dq2;
            else
                *qmax = dq0;

            *qmin = dq0 + dq1;
            if (*qmin >= 0.0) *qmin = 0.0;
        }
    } else {//dq0<0
        if (dq1 <= dq2) {
            if (dq1 < 0.0)
                *qmin = dq0 + dq1;
            else
                *qmin = dq0;

            *qmax = dq0 + dq2;
            if (*qmax <= 0.0) *qmax = 0.0;
        } else {// dq1>dq2
            if (dq2 < 0.0)
                *qmin = dq0 + dq2;
            else
                *qmin = dq0;

            *qmax = dq0 + dq1;
            if (*qmax <= 0.0) *qmax = 0.0;
        }
    }
    return 0;
}


#define Eepsilon                0
#define Eminimum_allowed_height 1
#define Ebeta_w                 2
#define Ebeta_w_dry             3
#define Ebeta_uh                4
#define Ebeta_uh_dry            5
#define Ebeta_vh                6
#define Ebeta_vh_dry            7
#define Eoptimise_dry_cells     8

/*
 * elements[0] = _epsilon
 * elements[1] = _minimum_allowed_height
 * elements[2] = _beta_w
 * elements[3] = _beta_w_dry
 * elements[4] = _beta_uh
 * elements[5] = _beta_uh_dry
 * elements[6] = _beta_vh
 * elements[7] = _beta_vh_dry
 * elements[8] = _optimise_dry_cells
 */


__global__ void _extrapolate_second_order_sw_TRUE(
        double * elements,
        long * surrogate_neighbours,
        long * number_of_boundaries,
        double * centroid_coordinates,
        double * bed_centroid_coordinates,
        double * bed_centroid_values,
        double * stage_centroid_values,
        double * xmom_centroid_values,
        double * ymom_centroid_values,
        double * elevation_centroid_values,
        double * vertex_coordinates,
        double * bed_vertex_values,
        double * stage_vertex_values,
        double * xmom_vertex_values,
        double * ymom_vertex_values,
        double * elevation_vertex_values,

        double * xmom_centroid_store, 
        double * ymom_centroid_store, 
        double * stage_centroid_store)
{
    int k = threadIdx.x + threadIdx.y + blockIdx.x * blockDim.x + blockIdx.y *blockDim.y;

    // Local variables
    double a, b; // Gradient vector used to calculate edge values from centroids
    int k0, k1, k2, k3, k6, coord_index, i;
    double x, y, x0, y0, x1, y1, x2, y2, xv0, yv0, xv1, yv1, xv2, yv2; // Vertices of the auxiliary triangle
    double dx1, dx2, dy1, dy2, dxv0, dxv1, dxv2, dyv0, dyv1, dyv2, dq0, dq1, dq2, area2, inv_area2;
    double dqv[3], qmin, qmax, hmin, hmax;
    double hc, h0, h1, h2, beta_tmp, hfactor;
    //double dk, dv0, dv1, dv2, de[3], demin, dcmax, r0scale;
    double dk, dv0, dv1, dv2;


    //weng_true if (extrapolate_velocity_second_order == 1) {
    // Replace momentum centroid with velocity centroid to allow velocity
    // extrapolation This will be changed back at the end of the routine
    //weng_true for (k = 0; k < number_of_elements; k++) {

    dk = max(stage_centroid_values[k] - bed_centroid_values[k], elements[Eminimum_allowed_height]);
    xmom_centroid_store[k] = xmom_centroid_values[k];
    xmom_centroid_values[k] = xmom_centroid_values[k] / dk;

    ymom_centroid_store[k] = ymom_centroid_values[k];
    ymom_centroid_values[k] = ymom_centroid_values[k] / dk;
    //weng_true }
    //weng_true }

    // Begin extrapolation routine
    //weng_true for (k = 0; k < number_of_elements; k++) {
    k3 = k * 3;
    k6 = k * 6;

    if (number_of_boundaries[k] == 3) {
        // No neighbours, set gradient on the triangle to zero

        stage_vertex_values[k3] = stage_centroid_values[k];
        stage_vertex_values[k3 + 1] = stage_centroid_values[k];
        stage_vertex_values[k3 + 2] = stage_centroid_values[k];
        xmom_vertex_values[k3] = xmom_centroid_values[k];
        xmom_vertex_values[k3 + 1] = xmom_centroid_values[k];
        xmom_vertex_values[k3 + 2] = xmom_centroid_values[k];
        ymom_vertex_values[k3] = ymom_centroid_values[k];
        ymom_vertex_values[k3 + 1] = ymom_centroid_values[k];
        ymom_vertex_values[k3 + 2] = ymom_centroid_values[k];

        //continue;
        return;
    } else {
        // Triangle k has one or more neighbours.
        // Get centroid and vertex coordinates of the triangle

        // Get the vertex coordinates
        xv0 = vertex_coordinates[k6];
        yv0 = vertex_coordinates[k6 + 1];
        xv1 = vertex_coordinates[k6 + 2];
        yv1 = vertex_coordinates[k6 + 3];
        xv2 = vertex_coordinates[k6 + 4];
        yv2 = vertex_coordinates[k6 + 5];

        // Get the centroid coordinates
        coord_index = 2 * k;
        x = centroid_coordinates[coord_index];
        y = centroid_coordinates[coord_index + 1];

        // Store x- and y- differentials for the vertices of
        // triangle k relative to the centroid
        dxv0 = xv0 - x;
        dxv1 = xv1 - x;
        dxv2 = xv2 - x;
        dyv0 = yv0 - y;
        dyv1 = yv1 - y;
        dyv2 = yv2 - y;
    }




    if (number_of_boundaries[k] <= 1) {
        //==============================================
        // Number of boundaries <= 1
        //==============================================


        // If no boundaries, auxiliary triangle is formed
        // from the centroids of the three neighbours
        // If one boundary, auxiliary triangle is formed
        // from this centroid and its two neighbours

        k0 = surrogate_neighbours[k3];
        k1 = surrogate_neighbours[k3 + 1];
        k2 = surrogate_neighbours[k3 + 2];

        // Get the auxiliary triangle's vertex coordinates
        // (really the centroids of neighbouring triangles)
        coord_index = 2 * k0;
        x0 = centroid_coordinates[coord_index];
        y0 = centroid_coordinates[coord_index + 1];

        coord_index = 2 * k1;
        x1 = centroid_coordinates[coord_index];
        y1 = centroid_coordinates[coord_index + 1];

        coord_index = 2 * k2;
        x2 = centroid_coordinates[coord_index];
        y2 = centroid_coordinates[coord_index + 1];

        // Store x- and y- differentials for the vertices
        // of the auxiliary triangle
        dx1 = x1 - x0;
        dx2 = x2 - x0;
        dy1 = y1 - y0;
        dy2 = y2 - y0;

        // Calculate 2*area of the auxiliary triangle
        // The triangle is guaranteed to be counter-clockwise
        area2 = dy2 * dx1 - dy1*dx2;

        // If the mesh is 'weird' near the boundary,
        // the triangle might be flat or clockwise
        // Default to zero gradient
        if (area2 <= 0) {
            //printf("Error negative triangle area \n");
            //report_python_error(AT, "Negative triangle area");
            //return -1;

            stage_vertex_values[k3] = stage_centroid_values[k];
            stage_vertex_values[k3 + 1] = stage_centroid_values[k];
            stage_vertex_values[k3 + 2] = stage_centroid_values[k];
            xmom_vertex_values[k3] = xmom_centroid_values[k];
            xmom_vertex_values[k3 + 1] = xmom_centroid_values[k];
            xmom_vertex_values[k3 + 2] = xmom_centroid_values[k];
            ymom_vertex_values[k3] = ymom_centroid_values[k];
            ymom_vertex_values[k3 + 1] = ymom_centroid_values[k];
            ymom_vertex_values[k3 + 2] = ymom_centroid_values[k];

            //continue;
            return;
        }

        // Calculate heights of neighbouring cells
        hc = stage_centroid_values[k] - bed_centroid_values[k];
        h0 = stage_centroid_values[k0] - bed_centroid_values[k0];
        h1 = stage_centroid_values[k1] - bed_centroid_values[k1];
        h2 = stage_centroid_values[k2] - bed_centroid_values[k2];
        hmin = min(min(h0, min(h1, h2)), hc);
        //hfactor = hc/(hc + 1.0);

        hfactor = 0.0;
        if (hmin > 0.001) {
            hfactor = (hmin - 0.001) / (hmin + 0.004);
        }

        if (elements[Eoptimise_dry_cells]) {
            // Check if linear reconstruction is necessary for triangle k
            // This check will exclude dry cells.

            hmax = max(h0, max(h1, h2));
            if (hmax < elements[Eepsilon]) {
                //continue;
                return;
            }
        }

        //-----------------------------------
        // stage
        //-----------------------------------

        // Calculate the difference between vertex 0 of the auxiliary
        // triangle and the centroid of triangle k
        dq0 = stage_centroid_values[k0] - stage_centroid_values[k];

        // Calculate differentials between the vertices
        // of the auxiliary triangle (centroids of neighbouring triangles)
        dq1 = stage_centroid_values[k1] - stage_centroid_values[k0];
        dq2 = stage_centroid_values[k2] - stage_centroid_values[k0];

        inv_area2 = 1.0 / area2;
        // Calculate the gradient of stage on the auxiliary triangle
        a = dy2 * dq1 - dy1*dq2;
        a *= inv_area2;
        b = dx1 * dq2 - dx2*dq1;
        b *= inv_area2;

        // Calculate provisional jumps in stage from the centroid
        // of triangle k to its vertices, to be limited
        dqv[0] = a * dxv0 + b*dyv0;
        dqv[1] = a * dxv1 + b*dyv1;
        dqv[2] = a * dxv2 + b*dyv2;

        // Now we want to find min and max of the centroid and the
        // vertices of the auxiliary triangle and compute jumps
        // from the centroid to the min and max
        find_qmin_and_qmax(dq0, dq1, dq2, &qmin, &qmax);

        // Playing with dry wet interface
        //hmin = qmin;
        //beta_tmp = beta_w_dry;
        //if (hmin>minimum_allowed_height)
        beta_tmp = elements[Ebeta_w_dry] + (elements[Ebeta_w] - elements[Ebeta_w_dry]) * hfactor;

        //printf("min_alled_height = %f\n",minimum_allowed_height);
        //printf("hmin = %f\n",hmin);
        //printf("beta_w = %f\n",beta_w);
        //printf("beta_tmp = %f\n",beta_tmp);
        // Limit the gradient
        limit_gradient(dqv, qmin, qmax, beta_tmp);

        //for (i=0;i<3;i++)
        stage_vertex_values[k3 + 0] = stage_centroid_values[k] + dqv[0];
        stage_vertex_values[k3 + 1] = stage_centroid_values[k] + dqv[1];
        stage_vertex_values[k3 + 2] = stage_centroid_values[k] + dqv[2];


        //-----------------------------------
        // xmomentum
        //-----------------------------------

        // Calculate the difference between vertex 0 of the auxiliary
        // triangle and the centroid of triangle k
        dq0 = xmom_centroid_values[k0] - xmom_centroid_values[k];

        // Calculate differentials between the vertices
        // of the auxiliary triangle
        dq1 = xmom_centroid_values[k1] - xmom_centroid_values[k0];
        dq2 = xmom_centroid_values[k2] - xmom_centroid_values[k0];

        // Calculate the gradient of xmom on the auxiliary triangle
        a = dy2 * dq1 - dy1*dq2;
        a *= inv_area2;
        b = dx1 * dq2 - dx2*dq1;
        b *= inv_area2;

        // Calculate provisional jumps in stage from the centroid
        // of triangle k to its vertices, to be limited
        dqv[0] = a * dxv0 + b*dyv0;
        dqv[1] = a * dxv1 + b*dyv1;
        dqv[2] = a * dxv2 + b*dyv2;

        // Now we want to find min and max of the centroid and the
        // vertices of the auxiliary triangle and compute jumps
        // from the centroid to the min and max
        find_qmin_and_qmax(dq0, dq1, dq2, &qmin, &qmax);
        //beta_tmp = beta_uh;
        //if (hmin<minimum_allowed_height)
        //beta_tmp = beta_uh_dry;
        beta_tmp = elements[Ebeta_uh_dry] + (elements[Ebeta_uh] - elements[Ebeta_uh_dry]) * hfactor;

        // Limit the gradient
        limit_gradient(dqv, qmin, qmax, beta_tmp);

        for (i = 0; i < 3; i++) {
            xmom_vertex_values[k3 + i] = xmom_centroid_values[k] + dqv[i];
        }

        //-----------------------------------
        // ymomentum
        //-----------------------------------

        // Calculate the difference between vertex 0 of the auxiliary
        // triangle and the centroid of triangle k
        dq0 = ymom_centroid_values[k0] - ymom_centroid_values[k];

        // Calculate differentials between the vertices
        // of the auxiliary triangle
        dq1 = ymom_centroid_values[k1] - ymom_centroid_values[k0];
        dq2 = ymom_centroid_values[k2] - ymom_centroid_values[k0];

        // Calculate the gradient of xmom on the auxiliary triangle
        a = dy2 * dq1 - dy1*dq2;
        a *= inv_area2;
        b = dx1 * dq2 - dx2*dq1;
        b *= inv_area2;

        // Calculate provisional jumps in stage from the centroid
        // of triangle k to its vertices, to be limited
        dqv[0] = a * dxv0 + b*dyv0;
        dqv[1] = a * dxv1 + b*dyv1;
        dqv[2] = a * dxv2 + b*dyv2;

        // Now we want to find min and max of the centroid and the
        // vertices of the auxiliary triangle and compute jumps
        // from the centroid to the min and max
        find_qmin_and_qmax(dq0, dq1, dq2, &qmin, &qmax);

        //beta_tmp = beta_vh;
        //
        //if (hmin<minimum_allowed_height)
        //beta_tmp = beta_vh_dry;
        beta_tmp = elements[Ebeta_vh_dry] + (elements[Ebeta_vh] - elements[Ebeta_vh_dry]) * hfactor;

        // Limit the gradient
        limit_gradient(dqv, qmin, qmax, beta_tmp);

        for (i = 0; i < 3; i++) {
            ymom_vertex_values[k3 + i] = ymom_centroid_values[k] + dqv[i];
        }
    }// End number_of_boundaries <=1
    else {

        //==============================================
        // Number of boundaries == 2
        //==============================================

        // One internal neighbour and gradient is in direction of the neighbour's centroid

        // Find the only internal neighbour (k1?)
        for (k2 = k3; k2 < k3 + 3; k2++) {
            // Find internal neighbour of triangle k
            // k2 indexes the edges of triangle k

            if (surrogate_neighbours[k2] != k) {
                break;
            }
        }

        if ((k2 == k3 + 3)) {
            // If we didn't find an internal neighbour
            //report_python_error(AT, "Internal neighbour not found");
            //return -1;
            return;
        }

        k1 = surrogate_neighbours[k2];

        // The coordinates of the triangle are already (x,y).
        // Get centroid of the neighbour (x1,y1)
        coord_index = 2 * k1;
        x1 = centroid_coordinates[coord_index];
        y1 = centroid_coordinates[coord_index + 1];

        // Compute x- and y- distances between the centroid of
        // triangle k and that of its neighbour
        dx1 = x1 - x;
        dy1 = y1 - y;

        // Set area2 as the square of the distance
        area2 = dx1 * dx1 + dy1*dy1;

        // Set dx2=(x1-x0)/((x1-x0)^2+(y1-y0)^2)
        // and dy2=(y1-y0)/((x1-x0)^2+(y1-y0)^2) which
        // respectively correspond to the x- and y- gradients
        // of the conserved quantities
        dx2 = 1.0 / area2;
        dy2 = dx2*dy1;
        dx2 *= dx1;


        //-----------------------------------
        // stage
        //-----------------------------------

        // Compute differentials
        dq1 = stage_centroid_values[k1] - stage_centroid_values[k];

        // Calculate the gradient between the centroid of triangle k
        // and that of its neighbour
        a = dq1*dx2;
        b = dq1*dy2;

        // Calculate provisional vertex jumps, to be limited
        dqv[0] = a * dxv0 + b*dyv0;
        dqv[1] = a * dxv1 + b*dyv1;
        dqv[2] = a * dxv2 + b*dyv2;

        // Now limit the jumps
        if (dq1 >= 0.0) {
            qmin = 0.0;
            qmax = dq1;
        } else {
            qmin = dq1;
            qmax = 0.0;
        }

        // Limit the gradient
        limit_gradient(dqv, qmin, qmax, elements[Ebeta_w]);

        //for (i=0; i < 3; i++)
        //{
        stage_vertex_values[k3] = stage_centroid_values[k] + dqv[0];
        stage_vertex_values[k3 + 1] = stage_centroid_values[k] + dqv[1];
        stage_vertex_values[k3 + 2] = stage_centroid_values[k] + dqv[2];
        //}

        //-----------------------------------
        // xmomentum
        //-----------------------------------

        // Compute differentials
        dq1 = xmom_centroid_values[k1] - xmom_centroid_values[k];

        // Calculate the gradient between the centroid of triangle k
        // and that of its neighbour
        a = dq1*dx2;
        b = dq1*dy2;

        // Calculate provisional vertex jumps, to be limited
        dqv[0] = a * dxv0 + b*dyv0;
        dqv[1] = a * dxv1 + b*dyv1;
        dqv[2] = a * dxv2 + b*dyv2;

        // Now limit the jumps
        if (dq1 >= 0.0) {
            qmin = 0.0;
            qmax = dq1;
        } else {
            qmin = dq1;
            qmax = 0.0;
        }

        // Limit the gradient
        limit_gradient(dqv, qmin, qmax, elements[Ebeta_w]);

        //for (i=0;i<3;i++)
        //xmom_vertex_values[k3] = xmom_centroid_values[k] + dqv[0];
        //xmom_vertex_values[k3 + 1] = xmom_centroid_values[k] + dqv[1];
        //xmom_vertex_values[k3 + 2] = xmom_centroid_values[k] + dqv[2];

        for (i = 0; i < 3; i++) {
            xmom_vertex_values[k3 + i] = xmom_centroid_values[k] + dqv[i];
        }

        //-----------------------------------
        // ymomentum
        //-----------------------------------

        // Compute differentials
        dq1 = ymom_centroid_values[k1] - ymom_centroid_values[k];

        // Calculate the gradient between the centroid of triangle k
        // and that of its neighbour
        a = dq1*dx2;
        b = dq1*dy2;

        // Calculate provisional vertex jumps, to be limited
        dqv[0] = a * dxv0 + b*dyv0;
        dqv[1] = a * dxv1 + b*dyv1;
        dqv[2] = a * dxv2 + b*dyv2;

        // Now limit the jumps
        if (dq1 >= 0.0) {
            qmin = 0.0;
            qmax = dq1;
        }
        else {
            qmin = dq1;
            qmax = 0.0;
        }

        // Limit the gradient
        limit_gradient(dqv, qmin, qmax, elements[Ebeta_w]);

        //for (i=0;i<3;i++)
        //ymom_vertex_values[k3] = ymom_centroid_values[k] + dqv[0];
        //ymom_vertex_values[k3 + 1] = ymom_centroid_values[k] + dqv[1];
        //ymom_vertex_values[k3 + 2] = ymom_centroid_values[k] + dqv[2];

        for (i = 0; i < 3; i++) {
            ymom_vertex_values[k3 + i] = ymom_centroid_values[k] + dqv[i];
        }
        //ymom_vertex_values[k3] = ymom_centroid_values[k] + dqv[0];
        //ymom_vertex_values[k3 + 1] = ymom_centroid_values[k] + dqv[1];
        //ymom_vertex_values[k3 + 2] = ymom_centroid_values[k] + dqv[2];
    } // else [number_of_boundaries==2]




    //weng_true } // for k=0 to number_of_elements-1

    //weng_true if (extrapolate_velocity_second_order == 1) {
    // Convert back from velocity to momentum
    //weng_true for (k = 0; k < number_of_elements; k++) {
    k3 = 3 * k;
    //dv0 = max(stage_vertex_values[k3]-bed_vertex_values[k3],minimum_allowed_height);
    //dv1 = max(stage_vertex_values[k3+1]-bed_vertex_values[k3+1],minimum_allowed_height);
    //dv2 = max(stage_vertex_values[k3+2]-bed_vertex_values[k3+2],minimum_allowed_height);
    dv0 = max(stage_vertex_values[k3] - bed_vertex_values[k3], 0.);
    dv1 = max(stage_vertex_values[k3 + 1] - bed_vertex_values[k3 + 1], 0.);
    dv2 = max(stage_vertex_values[k3 + 2] - bed_vertex_values[k3 + 2], 0.);

    //Correct centroid and vertex values
    xmom_centroid_values[k] = xmom_centroid_store[k];
    xmom_vertex_values[k3] = xmom_vertex_values[k3] * dv0;
    xmom_vertex_values[k3 + 1] = xmom_vertex_values[k3 + 1] * dv1;
    xmom_vertex_values[k3 + 2] = xmom_vertex_values[k3 + 2] * dv2;

    ymom_centroid_values[k] = ymom_centroid_store[k];
    ymom_vertex_values[k3] = ymom_vertex_values[k3] * dv0;
    ymom_vertex_values[k3 + 1] = ymom_vertex_values[k3 + 1] * dv1;
    ymom_vertex_values[k3 + 2] = ymom_vertex_values[k3 + 2] * dv2;

    //weng_true }
    //weng_true }

}




/*
 * elements[0] = _epsilon
 * elements[1] = _minimum_allowed_height
 * elements[2] = _beta_w
 * elements[3] = _beta_w_dry
 * elements[4] = _beta_uh
 * elements[5] = _beta_uh_dry
 * elements[6] = _beta_vh
 * elements[7] = _beta_vh_dry
 * elements[8] = _optimise_dry_cells
 */


// Computational routine
//__global__ int _extrapolate_second_order_edge_sw(struct domain *D) 
__global__ void _extrapolate_second_order_edge_sw_TRUE(
        double * elements,
        long * surrogate_neighbours,
        long * number_of_boundaries,
        double * centroid_coordinates,
        double * stage_centroid_values,
        double * xmom_centroid_values,
        double * ymom_centroid_values,
        double * bed_centroid_values,
        double * vertex_coordinates,
        double * bed_vertex_values,
        double * stage_vertex_values,
        double * xmom_vertex_values,
        double * ymom_vertex_values,
        double * elevation_vertex_values,
        double * edge_coordinates,
        double * bed_edge_values,
        double * stage_edge_values,
        double * xmom_edge_values,
        double * ymom_edge_values,
        double * xmom_centroid_store, 
        double * ymom_centroid_store,
        double * stage_centroid_store)
{
    // Local variables
    int k = threadIdx.x + threadIdx.y + blockIdx.x * blockDim.x + blockIdx.y *blockDim.y;

    double a, b; // Gradient vector used to calculate edge values from centroids
    int k0, k1, k2, k3, k6, coord_index, i;
    double x, y, x0, y0, x1, y1, x2, y2, xv0, yv0, xv1, yv1, xv2, yv2; // Vertices of the auxiliary triangle
    double dx1, dx2, dy1, dy2, dxv0, dxv1, dxv2, dyv0, dyv1, dyv2, dq0, dq1, dq2, area2, inv_area2;
    double dqv[3], qmin, qmax, hmin, hmax;
    double hc, h0, h1, h2, beta_tmp, hfactor;
    //double dk, dv0, dv1, dv2, de[3], demin, dcmax, r0scale;
    double dk, de[3];



    //weng_true if(extrapolate_velocity_second_order==1){
    // Replace momentum centroid with velocity centroid to allow velocity
    // extrapolation This will be changed back at the end of the routine
    //weng_true for (k=0; k<number_of_elements; k++){

    dk = max(stage_centroid_values[k]-bed_centroid_values[k], elements[Eminimum_allowed_height]);
    xmom_centroid_store[k] = xmom_centroid_values[k];
    xmom_centroid_values[k] = xmom_centroid_values[k]/dk;

    ymom_centroid_store[k] = ymom_centroid_values[k];
    ymom_centroid_values[k] = ymom_centroid_values[k]/dk;

    //weng_true }
    //weng_true }

    // Begin extrapolation routine
    //weng_true for (k = 0; k < number_of_elements; k++){
    k3=k*3;
    k6=k*6;

    if (number_of_boundaries[k]==3)
        //if (0==0)
    {
        // No neighbours, set gradient on the triangle to zero

        stage_edge_values[k3]   = stage_centroid_values[k];
        stage_edge_values[k3+1] = stage_centroid_values[k];
        stage_edge_values[k3+2] = stage_centroid_values[k];
        xmom_edge_values[k3]    = xmom_centroid_values[k];
        xmom_edge_values[k3+1]  = xmom_centroid_values[k];
        xmom_edge_values[k3+2]  = xmom_centroid_values[k];
        ymom_edge_values[k3]    = ymom_centroid_values[k];
        ymom_edge_values[k3+1]  = ymom_centroid_values[k];
        ymom_edge_values[k3+2]  = ymom_centroid_values[k];

        //continue;
        return;
    }
    else
    {
        // Triangle k has one or more neighbours.
        // Get centroid and edge coordinates of the triangle

        // Get the edge coordinates
        xv0 = edge_coordinates[k6];
        yv0 = edge_coordinates[k6+1];
        xv1 = edge_coordinates[k6+2];
        yv1 = edge_coordinates[k6+3];
        xv2 = edge_coordinates[k6+4];
        yv2 = edge_coordinates[k6+5];

        // Get the centroid coordinates
        coord_index = 2*k;
        x = centroid_coordinates[coord_index];
        y = centroid_coordinates[coord_index+1];

        // Store x- and y- differentials for the edges of
        // triangle k relative to the centroid
        dxv0 = xv0 - x;
        dxv1 = xv1 - x;
        dxv2 = xv2 - x;
        dyv0 = yv0 - y;
        dyv1 = yv1 - y;
        dyv2 = yv2 - y;
        // Compute the minimum distance from the centroid to an edge
        //demin=min(dxv0*dxv0 +dyv0*dyv0, min(dxv1*dxv1+dyv1*dyv1, dxv2*dxv2+dyv2*dyv2));
        //demin=sqrt(demin);
    }



    if (number_of_boundaries[k]<=1)
    {
        //==============================================
        // Number of boundaries <= 1
        //==============================================


        // If no boundaries, auxiliary triangle is formed
        // from the centroids of the three neighbours
        // If one boundary, auxiliary triangle is formed
        // from this centroid and its two neighbours

        k0 = surrogate_neighbours[k3];
        k1 = surrogate_neighbours[k3 + 1];
        k2 = surrogate_neighbours[k3 + 2];

        // Test to see whether we accept the surrogate neighbours
        // Note that if ki is replaced with k in more than 1 neighbour, then the
        // triangle area will be zero, and a first order extrapolation will be
        // used
        if(stage_centroid_values[k2]<=bed_centroid_values[k2]){
            k2 = k ;
        }
        if(stage_centroid_values[k0]<=bed_centroid_values[k0]){
            k0 = k ;
        }
        if(stage_centroid_values[k1]<=bed_centroid_values[k1]){
            k1 = k ;
        }
        // Alternative approach (BRUTAL) -- if the max neighbour bed elevation is greater
        // than the min neighbour stage, then we use first order extrapolation
        //bedmax = max(bed_centroid_values[k],
        //             max(bed_centroid_values[k0],
        //                 max(bed_centroid_values[k1], bed_centroid_values[k2])));
        //stagemin = min(stage_centroid_values[k],
        //             min(stage_centroid_values[k0],
        //                 min(stage_centroid_values[k1], stage_centroid_values[k2])));
        //
        //if(stagemin < bedmax){
        //   // This will cause first order extrapolation
        //   k2 = k;
        //   k0 = k;
        //   k1 = k;
        //}

        // Get the auxiliary triangle's vertex coordinates
        // (really the centroids of neighbouring triangles)
        coord_index = 2*k0;
        x0 = centroid_coordinates[coord_index];
        y0 = centroid_coordinates[coord_index+1];

        coord_index = 2*k1;
        x1 = centroid_coordinates[coord_index];
        y1 = centroid_coordinates[coord_index+1];

        coord_index = 2*k2;
        x2 = centroid_coordinates[coord_index];
        y2 = centroid_coordinates[coord_index+1];

        // compute the maximum distance from the centroid to a neighbouring
        // centroid
        //dcmax=max( (x0-x)*(x0-x) + (y0-y)*(y0-y),
        //           max((x1-x)*(x1-x) + (y1-y)*(y1-y),
        //               (x2-x)*(x2-x) + (y2-y)*(y2-y)));
        //dcmax=sqrt(dcmax);
        //// Ratio of centroid to edge distance -- useful in attempting to adapt limiter
        //if(dcmax>0.){
        //    r0scale=demin/dcmax;
        //    //printf("%f \n", r0scale);
        //}else{
        //    r0scale=0.5;
        //}

        // Store x- and y- differentials for the vertices
        // of the auxiliary triangle
        dx1 = x1 - x0;
        dx2 = x2 - x0;
        dy1 = y1 - y0;
        dy2 = y2 - y0;

        // Calculate 2*area of the auxiliary triangle
        // The triangle is guaranteed to be counter-clockwise
        area2 = dy2*dx1 - dy1*dx2;

        // If the mesh is 'weird' near the boundary,
        // the triangle might be flat or clockwise
        // Default to zero gradient
        if (area2 <= 0)
        {
            //printf("Error negative triangle area \n");
            //report_python_error(AT, "Negative triangle area");
            //return -1;

            stage_edge_values[k3]   = stage_centroid_values[k];
            stage_edge_values[k3+1] = stage_centroid_values[k];
            stage_edge_values[k3+2] = stage_centroid_values[k];
            xmom_edge_values[k3]    = xmom_centroid_values[k];
            xmom_edge_values[k3+1]  = xmom_centroid_values[k];
            xmom_edge_values[k3+2]  = xmom_centroid_values[k];
            ymom_edge_values[k3]    = ymom_centroid_values[k];
            ymom_edge_values[k3+1]  = ymom_centroid_values[k];
            ymom_edge_values[k3+2]  = ymom_centroid_values[k];

            //continue;
            return;
        }

        // Calculate heights of neighbouring cells
        hc = stage_centroid_values[k]  - bed_centroid_values[k];
        h0 = stage_centroid_values[k0] - bed_centroid_values[k0];
        h1 = stage_centroid_values[k1] - bed_centroid_values[k1];
        h2 = stage_centroid_values[k2] - bed_centroid_values[k2];
        hmin = min(min(h0, min(h1, h2)), hc);
        //hmin = min(h0, min(h1, h2));
        //hmin = max(hmin, 0.0);
        //hfactor = hc/(hc + 1.0);

        hfactor = 0.0;
        //if (hmin > 0.001)
        if (hmin > 0.)
            //if (hc>0.0)
        {
            hfactor = 1.0 ;//hmin/(hmin + 0.004);
            //hfactor=hmin/(hmin + 0.004);
        }

        if (elements[Eoptimise_dry_cells])
        {
            // Check if linear reconstruction is necessary for triangle k
            // This check will exclude dry cells.

            //hmax = max(h0, max(h1, max(h2, hc)));
            hmax = max(h0, max(h1, h2));
            if (hmax < elements[Eepsilon])
            {
                //continue;
                return;
            }
        }

        //-----------------------------------
        // stage
        //-----------------------------------

        // Calculate the difference between vertex 0 of the auxiliary
        // triangle and the centroid of triangle k
        dq0 = stage_centroid_values[k0] - stage_centroid_values[k];

        // Calculate differentials between the vertices
        // of the auxiliary triangle (centroids of neighbouring triangles)
        dq1 = stage_centroid_values[k1] - stage_centroid_values[k0];
        dq2 = stage_centroid_values[k2] - stage_centroid_values[k0];

        inv_area2 = 1.0/area2;
        // Calculate the gradient of stage on the auxiliary triangle
        a = dy2*dq1 - dy1*dq2;
        a *= inv_area2;
        b = dx1*dq2 - dx2*dq1;
        b *= inv_area2;

        // Calculate provisional jumps in stage from the centroid
        // of triangle k to its vertices, to be limited
        dqv[0] = a*dxv0 + b*dyv0;
        dqv[1] = a*dxv1 + b*dyv1;
        dqv[2] = a*dxv2 + b*dyv2;

        // Now we want to find min and max of the centroid and the
        // vertices of the auxiliary triangle and compute jumps
        // from the centroid to the min and max
        find_qmin_and_qmax(dq0, dq1, dq2, &qmin, &qmax);

        beta_tmp = elements[Ebeta_w_dry] + (elements[Ebeta_w] - elements[Ebeta_w_dry]) * hfactor;

        // Limit the gradient
        limit_gradient(dqv, qmin, qmax, beta_tmp);
        //limit_gradient2(dqv, qmin, qmax, beta_tmp,r0scale);

        //for (i=0;i<3;i++)
        stage_edge_values[k3+0] = stage_centroid_values[k] + dqv[0];
        stage_edge_values[k3+1] = stage_centroid_values[k] + dqv[1];
        stage_edge_values[k3+2] = stage_centroid_values[k] + dqv[2];

        //-----------------------------------
        // xmomentum
        //-----------------------------------

        // Calculate the difference between vertex 0 of the auxiliary
        // triangle and the centroid of triangle k
        dq0 = xmom_centroid_values[k0] - xmom_centroid_values[k];

        // Calculate differentials between the vertices
        // of the auxiliary triangle
        dq1 = xmom_centroid_values[k1] - xmom_centroid_values[k0];
        dq2 = xmom_centroid_values[k2] - xmom_centroid_values[k0];

        // Calculate the gradient of xmom on the auxiliary triangle
        a = dy2*dq1 - dy1*dq2;
        a *= inv_area2;
        b = dx1*dq2 - dx2*dq1;
        b *= inv_area2;

        // Calculate provisional jumps in stage from the centroid
        // of triangle k to its vertices, to be limited
        dqv[0] = a*dxv0+b*dyv0;
        dqv[1] = a*dxv1+b*dyv1;
        dqv[2] = a*dxv2+b*dyv2;

        // Now we want to find min and max of the centroid and the
        // vertices of the auxiliary triangle and compute jumps
        // from the centroid to the min and max
        //
        find_qmin_and_qmax(dq0, dq1, dq2, &qmin, &qmax);

        beta_tmp = elements[Ebeta_uh_dry] + (elements[Ebeta_uh] - elements[Ebeta_uh_dry]) * hfactor;

        // Limit the gradient
        limit_gradient(dqv, qmin, qmax, beta_tmp);
        //limit_gradient2(dqv, qmin, qmax, beta_tmp,r0scale);


        for (i=0; i < 3; i++)
        {
            xmom_edge_values[k3+i] = xmom_centroid_values[k] + dqv[i];
        }

        //-----------------------------------
        // ymomentum
        //-----------------------------------

        // Calculate the difference between vertex 0 of the auxiliary
        // triangle and the centroid of triangle k
        dq0 = ymom_centroid_values[k0] - ymom_centroid_values[k];

        // Calculate differentials between the vertices
        // of the auxiliary triangle
        dq1 = ymom_centroid_values[k1] - ymom_centroid_values[k0];
        dq2 = ymom_centroid_values[k2] - ymom_centroid_values[k0];

        // Calculate the gradient of xmom on the auxiliary triangle
        a = dy2*dq1 - dy1*dq2;
        a *= inv_area2;
        b = dx1*dq2 - dx2*dq1;
        b *= inv_area2;

        // Calculate provisional jumps in stage from the centroid
        // of triangle k to its vertices, to be limited
        dqv[0] = a*dxv0 + b*dyv0;
        dqv[1] = a*dxv1 + b*dyv1;
        dqv[2] = a*dxv2 + b*dyv2;

        // Now we want to find min and max of the centroid and the
        // vertices of the auxiliary triangle and compute jumps
        // from the centroid to the min and max
        //
        find_qmin_and_qmax(dq0, dq1, dq2, &qmin, &qmax);

        beta_tmp = elements[Ebeta_vh_dry] + (elements[Ebeta_vh] - elements[Ebeta_vh_dry]) * hfactor;

        // Limit the gradient
        limit_gradient(dqv, qmin, qmax, beta_tmp);
        //limit_gradient2(dqv, qmin, qmax, beta_tmp,r0scale);

        for (i=0;i<3;i++)
        {
            ymom_edge_values[k3 + i] = ymom_centroid_values[k] + dqv[i];
        }

    } // End number_of_boundaries <=1
    else
    {

        //==============================================
        // Number of boundaries == 2
        //==============================================

        // One internal neighbour and gradient is in direction of the neighbour's centroid

        // Find the only internal neighbour (k1?)
        for (k2 = k3; k2 < k3 + 3; k2++)
        {
            // Find internal neighbour of triangle k
            // k2 indexes the edges of triangle k

            if (surrogate_neighbours[k2] != k)
            {
                break;
            }
        }

        if ((k2 == k3 + 3))
        {
            // If we didn't find an internal neighbour
            // report_python_error(AT, "Internal neighbour not found");
            //return -1;
            return;
        }

        k1 = surrogate_neighbours[k2];

        // The coordinates of the triangle are already (x,y).
        // Get centroid of the neighbour (x1,y1)
        coord_index = 2*k1;
        x1 = centroid_coordinates[coord_index];
        y1 = centroid_coordinates[coord_index + 1];

        // Compute x- and y- distances between the centroid of
        // triangle k and that of its neighbour
        dx1 = x1 - x;
        dy1 = y1 - y;

        // Set area2 as the square of the distance
        area2 = dx1*dx1 + dy1*dy1;

        // Set dx2=(x1-x0)/((x1-x0)^2+(y1-y0)^2)
        // and dy2=(y1-y0)/((x1-x0)^2+(y1-y0)^2) which
        // respectively correspond to the x- and y- gradients
        // of the conserved quantities
        dx2 = 1.0/area2;
        dy2 = dx2*dy1;
        dx2 *= dx1;


        //-----------------------------------
        // stage
        //-----------------------------------

        // Compute differentials
        dq1 = stage_centroid_values[k1] - stage_centroid_values[k];

        // Calculate the gradient between the centroid of triangle k
        // and that of its neighbour
        a = dq1*dx2;
        b = dq1*dy2;

        // Calculate provisional edge jumps, to be limited
        dqv[0] = a*dxv0 + b*dyv0;
        dqv[1] = a*dxv1 + b*dyv1;
        dqv[2] = a*dxv2 + b*dyv2;

        // Now limit the jumps
        if (dq1>=0.0)
        {
            qmin=0.0;
            qmax=dq1;
        }
        else
        {
            qmin = dq1;
            qmax = 0.0;
        }

        // Limit the gradient
        limit_gradient(dqv, qmin, qmax, elements[Ebeta_w]);

        //for (i=0; i < 3; i++)
        //{
        stage_edge_values[k3] = stage_centroid_values[k] + dqv[0];
        stage_edge_values[k3 + 1] = stage_centroid_values[k] + dqv[1];
        stage_edge_values[k3 + 2] = stage_centroid_values[k] + dqv[2];
        //}

        //-----------------------------------
        // xmomentum
        //-----------------------------------

        // Compute differentials
        dq1 = xmom_centroid_values[k1] - xmom_centroid_values[k];

        // Calculate the gradient between the centroid of triangle k
        // and that of its neighbour
        a = dq1*dx2;
        b = dq1*dy2;

        // Calculate provisional edge jumps, to be limited
        dqv[0] = a*dxv0+b*dyv0;
        dqv[1] = a*dxv1+b*dyv1;
        dqv[2] = a*dxv2+b*dyv2;

        // Now limit the jumps
        if (dq1 >= 0.0)
        {
            qmin = 0.0;
            qmax = dq1;
        }
        else
        {
            qmin = dq1;
            qmax = 0.0;
        }

        // Limit the gradient
        limit_gradient(dqv, qmin, qmax, elements[Ebeta_w]);

        //for (i=0;i<3;i++)
        //xmom_edge_values[k3] = xmom_centroid_values[k] + dqv[0];
        //xmom_edge_values[k3 + 1] = xmom_centroid_values[k] + dqv[1];
        //xmom_edge_values[k3 + 2] = xmom_centroid_values[k] + dqv[2];

        for (i = 0; i < 3;i++)
        {
            xmom_edge_values[k3 + i] = xmom_centroid_values[k] + dqv[i];
        }

        //-----------------------------------
        // ymomentum
        //-----------------------------------

        // Compute differentials
        dq1 = ymom_centroid_values[k1] - ymom_centroid_values[k];

        // Calculate the gradient between the centroid of triangle k
        // and that of its neighbour
        a = dq1*dx2;
        b = dq1*dy2;

        // Calculate provisional edge jumps, to be limited
        dqv[0] = a*dxv0 + b*dyv0;
        dqv[1] = a*dxv1 + b*dyv1;
        dqv[2] = a*dxv2 + b*dyv2;

        // Now limit the jumps
        if (dq1>=0.0)
        {
            qmin = 0.0;
            qmax = dq1;
        }
        else
        {
            qmin = dq1;
            qmax = 0.0;
        }

        // Limit the gradient
        limit_gradient(dqv, qmin, qmax, elements[Ebeta_w]);

        for (i=0;i<3;i++)
        {
            ymom_edge_values[k3 + i] = ymom_centroid_values[k] + dqv[i];
        }
    } // else [number_of_boundaries==2]
    //weng_true } // for k=0 to number_of_elements-1

    // Compute vertex values of quantities
    //weng_true for (k=0; k<number_of_elements; k++){
    k3 = 3*k;

    // Compute stage vertex values
    stage_vertex_values[k3] = stage_edge_values[k3+1] + stage_edge_values[k3+2] -stage_edge_values[k3] ;
    stage_vertex_values[k3+1] =  stage_edge_values[k3] + stage_edge_values[k3+2]-stage_edge_values[k3+1];
    stage_vertex_values[k3+2] =  stage_edge_values[k3] + stage_edge_values[k3+1]-stage_edge_values[k3+2];

    // Compute xmom vertex values
    xmom_vertex_values[k3] = xmom_edge_values[k3+1] + xmom_edge_values[k3+2] -xmom_edge_values[k3] ;
    xmom_vertex_values[k3+1] =  xmom_edge_values[k3] + xmom_edge_values[k3+2]-xmom_edge_values[k3+1];
    xmom_vertex_values[k3+2] =  xmom_edge_values[k3] + xmom_edge_values[k3+1]-xmom_edge_values[k3+2];

    // Compute ymom vertex values
    ymom_vertex_values[k3] = ymom_edge_values[k3+1] + ymom_edge_values[k3+2] -ymom_edge_values[k3] ;
    ymom_vertex_values[k3+1] =  ymom_edge_values[k3] + ymom_edge_values[k3+2]-ymom_edge_values[k3+1];
    ymom_vertex_values[k3+2] =  ymom_edge_values[k3] + ymom_edge_values[k3+1]-ymom_edge_values[k3+2];

    // If needed, convert from velocity to momenta
    //weng_true if(extrapolate_velocity_second_order==1){
    //Convert velocity back to momenta at centroids
    xmom_centroid_values[k] = xmom_centroid_store[k];
    ymom_centroid_values[k] = ymom_centroid_store[k];

    // Re-compute momenta at edges
    for (i=0; i<3; i++){
        de[i] = max(stage_edge_values[k3+i]-bed_edge_values[k3+i],0.0);
        xmom_edge_values[k3+i]=xmom_edge_values[k3+i]*de[i];
        ymom_edge_values[k3+i]=ymom_edge_values[k3+i]*de[i];
    }

    // Re-compute momenta at vertices
    for (i=0; i<3; i++){
        de[i] = max(stage_vertex_values[k3+i]-bed_vertex_values[k3+i],0.0);
        xmom_vertex_values[k3+i]=xmom_vertex_values[k3+i]*de[i];
        ymom_vertex_values[k3+i]=ymom_vertex_values[k3+i]*de[i];
    }
    //weng_true }
    //weng_true }
}


#ifndef MAIN_EXTRAPOLATE
#define MAIN_EXTRAPOLATE
int main()
{}
#endif
