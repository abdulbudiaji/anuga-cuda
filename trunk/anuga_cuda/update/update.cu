
#include <hip/hip_runtime.h>
// from quantity_ext.c
__global__ void update(
        int N,
        double timestep,
        double * centroid_values,
        double * explicit_update,
        double * semi_implicit_update)
{
    const int k = 
            threadIdx.x+threadIdx.y*blockDim.x+
            (blockIdx.x+blockIdx.y*gridDim.x)*blockDim.x*blockDim.y;

    double denominator, x;

    if (k >= N)
        return;

    // Divide semi_implicit update by conserved quantity
    //for (k=0; k<N; k++) {
    x = centroid_values[k];
    if (x == 0.0) {
        semi_implicit_update[k] = 0.0;
    } else {
        semi_implicit_update[k] /= x;
    }
    //}


    // Explicit updates
    //for (k=0; k<N; k++) {
    centroid_values[k] += timestep*explicit_update[k];
    //}



    // Semi implicit updates
    //for (k=0; k<N; k++) {
    denominator = 1.0 - timestep*semi_implicit_update[k];
    if (denominator <= 0.0) {
        return;
    } else {
        //Update conserved_quantities from semi implicit updates
        centroid_values[k] /= denominator;
    }
    //}



    // Reset semi_implicit_update here ready for next time step
    //memset(semi_implicit_update, 0, N*sizeof(double));
}

